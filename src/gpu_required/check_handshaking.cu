/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2021                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void check_handshaking_gpu(int * strongNeighbor, int * matches, int numNodes) {
	/** YOUR CODE GOES BELOW **/
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    for(int node = tid; node < numNodes; node += numThreads) {
        int strongestNeighbor = strongNeighbor[node];
        if(strongestNeighbor < 0 || matches[node] >= 0) continue;
        if(strongNeighbor[strongestNeighbor] == node) {
            matches[node] = strongestNeighbor;
            matches[strongestNeighbor] = node;
        }
    }
	/** YOUR CODE GOES ABOVE **/
}
