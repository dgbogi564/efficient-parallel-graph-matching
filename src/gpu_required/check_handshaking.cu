#include "hip/hip_runtime.h"
/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2021                                 *
 **********************************************
 */
#include <stdio.h>
#include <stdlib.h>
#include "utils.hpp"

__global__ void check_handshaking_gpu(int * strongNeighbor, int * matches, int numNodes) {
	/** YOUR CODE GOES BELOW **/
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    for(int node = tid; node < numNodes; node += blockDim.x * gridDim.x) {
        int strongestNeighbor = strongNeighbor[node];
        if(strongestNeighbor < 0 || matches[node] >= 0) continue;
        if(strongNeighbor[strongestNeighbor] == node) {
            matches[node] = strongestNeighbor;
            matches[strongestNeighbor] = node;
        }
    }
	/** YOUR CODE GOES ABOVE **/
}
