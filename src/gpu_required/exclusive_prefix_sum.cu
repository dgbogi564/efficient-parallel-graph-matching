#include "hip/hip_runtime.h"
/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2021                                 *
 **********************************************
 */
#include <stdio.h>
#include <stdlib.h>
#include <utils.hpp>

__global__ void exclusive_prefix_sum_gpu(int * oldSum, int * newSum, int distance, int numElements) {
	/** YOUR CODE GOES BELOW **/
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    int x = 0;
    if(!distance) {
        for(int element = tid; element < numElements-1; element += numThreads) {
            newSum[element+1] = oldSum[element];
        }
        if(tid == 0) {
            oldSum[0] = 0;
            newSum[0] = 0;
        }
    } else {
        for(int element = tid; element < numElements; element += numThreads) {
            if(element < distance) newSum[element] = oldSum[element];
            else newSum[element] = oldSum[element-distance] + oldSum[element];
        }
    }
    /** YOUR CODE GOES ABOVE **/
}
