/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2021                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void markFilterEdges_gpu(int * src, int * dst, int * matches, int * keepEdges, int numEdges) {
	/** YOUR CODE GOES BELOW **/
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    for(int edge = tid; edge < numEdges; edge += numThreads) {
        keepEdges[edge] = matches[src[edge]] < 0 && matches[dst[edge]] < 0;
    }
	/** YOUR CODE GOES ABOVE **/
}
